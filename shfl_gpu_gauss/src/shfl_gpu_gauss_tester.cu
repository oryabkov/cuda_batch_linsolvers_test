#include "hip/hip_runtime.h"

#include <cstdlib>
#include <cstdio>
#include <cmath>
#include <iostream>
#include <string>
#include <stdexcept>
#include <boost/program_options.hpp>
#include <boost/algorithm/string.hpp>
#include <hip/hip_runtime.h>
#include <hip/hip_runtime.h>

#include "cuda_timer_event.h"
#include "cuda_safe_call.h"
#include "index_matrix.h"
#include "matrix_utils.h"
#include "init_dense_sz.h"
#include "copy_dense_to_dense.h"
#include "copy_sparse_to_dense.h"
#include "write_vector.h"
#include "shfl_gpu_gauss.cuh"

namespace po = boost::program_options;

typedef SCALAR_TYPE     real;

int main(int argc, char **args)
{
    std::string         input_path_A, input_path_b, output_path_x;
    int                 repeat_times;
    int                 device_number;

    try {
        std::cout << "Use -h option for help" << std::endl;

        po::options_description desc("Tester options");
        desc.add_options()
            ("help,h", "Show help")
            ("INPUT_A,a", po::value<std::string>()->default_value("big_A.mm"), "Input .mm file with A matrices")
            ("INPUT_B,b", po::value<std::string>()->default_value("b_vector.csv"), "Input .csv file with b vector")
            ("output,o", po::value<std::string>()->default_value("x_vector.csv"), "Output solution file")
            ("device_number,d", po::value<int>()->default_value(0), "Device number")
            ("repeat_times,r", po::value<int>()->default_value(10), "Number ot test repeats");

        po::positional_options_description desc_pos;

        po::variables_map   vm;
        po::store(po::command_line_parser(argc,args).options(desc).positional(desc_pos).run(), vm);
        po::notify(vm);

        if (vm.count("help")) {
            std::cout << desc << std::endl;
            return 0;
        }

        input_path_A = vm["INPUT_A"].as<std::string>();
        input_path_b = vm["INPUT_B"].as<std::string>();
        output_path_x = vm["output"].as<std::string>();

        device_number = vm["device_number"].as<int>();
        repeat_times = vm["repeat_times"].as<int>();
    } catch(std::exception& ex) {
        std::cerr << ex.what() << std::endl;
        return 1;
    }

    if (sizeof(real) == sizeof(float))
        std::cout << "Float variant is tested" << std::endl;
    else if (sizeof(real) == sizeof(double))
        std::cout << "Double variant is tested" << std::endl;
    else {
        std::cout << "Real is neither float nor double" << std::endl;
        return 1;
    }

    // struct for matrices
    batch_systems_data<real> batch_systems;
    int                      matrices_num_orig;

    try {
        // round up matrices num to block size
        read_matrices(input_path_A, input_path_b, batch_systems, matrices_num_orig, 256);
        std::cout << "Using rounded batch_size: " << batch_systems.matrices_num << std::endl;
        std::cout << "done" << std::endl;
    } catch(std::exception& ex) {
        std::cerr << "Error while reading matrices and rhs: " << ex.what() << std::endl;
        return 1;
    }

    /* if all ok - we have our matrices in structure batch_systems
     * let's print number of matrices, number of nnz elems and stats*/
    print_matrices_stats(batch_systems);

    std::cout << "Converting matrices to dense format..." << std::endl;
    // note that extended matrix is used so rhs is appended
    int     batch_sz, N, M;
    real    *matrices, *matrices_0;
    init_dense_sz(batch_systems, batch_sz, N, M);
    if (M > 32) {
        std::cout << "Number of columns (M) more than 32 is not supported by this solver" << std::endl;
        return 1;
    }
    CUDA_SAFE_CALL( hipHostMalloc((void**)&matrices, sizeof(real)*batch_sz*N*M) );
    CUDA_SAFE_CALL( hipHostMalloc((void**)&matrices_0, sizeof(real)*batch_sz*N*M) );
    copy_sparse_to_dense(batch_systems, batch_sz, N, M, matrices);
    copy_dense_to_dense(batch_sz, N, M, matrices, matrices_0);
    std::cout << "done" << std::endl;

    std::cout << "Initializating device number " << device_number << std::endl;
    hipDeviceProp_t deviceProp;
    hipGetDeviceProperties(&deviceProp, device_number);
    std::cout << "Device compute capability: " << deviceProp.major << "." << deviceProp.minor << std::endl;
    if (deviceProp.major*100 + deviceProp.minor < 305) {
        std::cout << "CC less then 3.5 is not supported by this solver" << std::endl;
        return 1;
    }
    hipSetDevice(device_number);
    std::cout << "done" << std::endl;

    real    *matrices_dev, *matrices_dev_0;

    std::cout << "Allocating device memory..." << std::endl;
    CUDA_SAFE_CALL( hipMalloc((void**)&matrices_dev, sizeof(real)*batch_sz*N*M) );
    CUDA_SAFE_CALL( hipMalloc((void**)&matrices_dev_0, sizeof(real)*batch_sz*N*M) );
    std::cout << "done" << std::endl;

    std::cout << "Copying data to gpu..." << std::endl;
    CUDA_SAFE_CALL( hipMemcpy(matrices_dev_0, matrices, sizeof(real)*batch_sz*N*M, hipMemcpyHostToDevice) );
    std::cout << "done" << std::endl;

    cuda_timer_event    start, end;
    start.init(); end.init();

    std::cout << "Calculation..." << std::endl;
    start.record();

    for (int iter = 0;iter < repeat_times;++iter) {
        shfl_gpu_gauss(batch_sz, N, M, matrices_dev_0, matrices_dev);
    }

    end.record();
    std::cout << "done" << std::endl;

    std::cout << "Elapsed time:           " << end.elapsed_time(start)/1000. << " s" << std::endl;
    std::cout << "Repeat times:           " << repeat_times << std::endl;
    std::cout << "Time per iteration:     " << end.elapsed_time(start)/1000./repeat_times << " s" << std::endl;

    std::cout << std::endl << "T1:" << repeat_times << "\t" << batch_sz << "\t" 
              << end.elapsed_time(start)/1000. << std::endl << std::endl;

    std::cout << "Copying results back to host..." << std::endl;

    CUDA_SAFE_CALL( hipMemcpy(matrices, matrices_dev, sizeof(real)*batch_sz*N*M, hipMemcpyDeviceToHost) );

    // we explititly use here that number of rhs's is 1
    std::cout << "Calculating residual on cpu..." << std::endl;
    real    norm_C = 0.f;
    for (int s = 0;s < batch_sz;++s) {
        for (int ii1 = 0;ii1 < N;++ii1) {
            real    res = 0.f;
            for (int ii2 = 0;ii2 < N;++ii2) {
                res += matrices_0[IM(s,ii1,ii2)]*matrices[IM(s,ii2,N)];
            }
            norm_C = fmax(fabs(res - matrices_0[IM(s,ii1,N)]), norm_C);
        }
    }
    std::cout << "done" << std::endl;
    std::cout << "Residual norm_C:        " << norm_C << std::endl;

    try {
        //write_vector(batch_sz, N, M, matrices, output_path_x);
        // NOTE use matrices_num_orig here instead of batch_sz to match input files shape
        write_vector(matrices_num_orig, N, M, matrices, output_path_x);
    } catch(std::exception& ex) {
        std::cerr << "Error while writing result: " << ex.what() << std::endl;
        return 1;
    }

    std::cout << "Free memory..." << std::endl;
    free_matrices(batch_systems);
    CUDA_SAFE_CALL( hipHostFree(matrices) );
    CUDA_SAFE_CALL( hipHostFree(matrices_0) );
    CUDA_SAFE_CALL( hipFree(matrices_dev) );
    CUDA_SAFE_CALL( hipFree(matrices_dev_0) );
    std::cout << "done" << std::endl;

    return 0;
}
